#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <iostream>

#include "simulation_gpu.h"

#define FULL_MASK 0xffffffff
#define TILE_DIM 51
#define TIMEPOINTS 51
#define VOL_DIM 3
#define BLOCKSIZE 32
#define WARPSIZE 32
#define MAX_BLOCK_SZ 256
#define BATCH_SZ 1000

#undef HJM_SDE_DEBUG
#define MC_RDM_DEBUG
#undef HJM_PATH_SIMULATION_DEBUG
#define HJM_NUMERAIRE_DEBUG
#undef EXPOSURE_PROFILES_DEBUG
#define DEV_CURND_HOSTGEN
#undef EXPOSURE_PROFILES_AGGR_DEBUG
#undef CONST_MEMORY
#define RNG_HOST_API
#undef RNG_DEV_API

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus)                                                      \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
    }


#define CURAND_CALL(x)                                 \
   {                                                   \
        if((x)!=HIPRAND_STATUS_SUCCESS)                 \
          printf("ERROR: CURAND call at %s:%d\n",__FILE__,__LINE__);\
                                                       \
    }  

#define CUBLAS_CALL(x)                                 \
   {                                                   \
        if((x)!=HIPBLAS_STATUS_SUCCESS)                 \
          printf("ERROR: CUBLAS call at %s:%d\n",__FILE__,__LINE__);\
                                                       \
    } 


/*
 * Musiela Parametrization SDE
 * We simulate the SDE f(t+dt)=f(t) + dfbar  
 * where SDE dfbar =  m(t)*dt+SUM(Vol_i*phi[i]*SQRT(dt))+dF/dtau*dt and phi ~ N(0,1)
 */

__device__
inline float __musiela_sde2(float drift, float vol0, float vol1, float vol2, float phi0, float phi1, float phi2, float sqrt_dt, float dF, float rate0, float dtau, float dt) {
    float vol_sum = vol0 * phi0;
    vol_sum += vol1 * phi1;
    vol_sum += vol2 * phi2;
    vol_sum *= sqrtf(dt);

    float dfbar = drift * dt;
    dfbar += vol_sum;

    dfbar += (dF / dtau) * dt;

    // apply Euler Maruyana
    float result = rate0 + dfbar;

    return result;
}

/*
 Calculate dF term in Musiela Parametrization SDE
*/

/*
__device__
inline float __dFau(int t, int timepoints, float* rates) {
    float result = 0.0;

    if (t == (timepoints - 1)) {
        result = rates[t] - rates[t - 1];
    }
    else {
        result = rates[t + 1] - rates[t];
    }

    return result;
}
*/


/*
 * Initialize auxiliary vectors used during the simulation
 */
__global__
void initVectors_kernel(float2* numeraires, float* simulated_rates, float* spot_rates, int exposuresCount, float dt) {
    int gindex = blockIdx.x* blockDim.x + threadIdx.x;
    int stride = TIMEPOINTS * gridDim.x;
    int N = TIMEPOINTS * exposuresCount;

    if ( (threadIdx.x < TIMEPOINTS) && (gindex < N) ) {
        float rate = spot_rates[threadIdx.x];
        float discount_factor = exp(-rate * dt);

        for (int i = gindex; i < N; i += stride) {
            // store the spot_rate
            simulated_rates[i] = rate;
            // initialize numeraire at t = 0
            if (threadIdx.x == 0) {
                numeraires[i].x = rate;
                numeraires[i].y = discount_factor;
            }
        }
    }
}


/**
* * RNG init Kernel
*/

#ifdef RNG_HOST_API
void initRNG2_kernel(float* rngNrmVar, const unsigned int seed, int rnd_count)
{
    const float mean = 0.0;  const float stddev = 1.0;
    hiprandGenerator_t generator;
    CURAND_CALL( hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT) );
    CURAND_CALL( hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL) );
    CURAND_CALL( hiprandGenerateNormal(generator, rngNrmVar, rnd_count, mean, stddev) );
    CUDA_RT_CALL( hipDeviceSynchronize() );
    CURAND_CALL(hiprandDestroyGenerator(generator));
}
#else
__global__ void initRNG2_kernel(hiprandStateMRG32k3a* const rngStates, const unsigned int seed, int rnd_count)
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    for (; index < rnd_count; index += blockDim.x * gridDim.x) {
        hiprand_init(seed, index, 0, &rngStates[index]);
    }
}
#endif

/*
 * Monte Carlo HJM Path Generation
*/
__global__
void __generatePaths_kernel(float2* numeraires, 
    float* drifts, float* volatilities, float* rngNrmVar,
    float* simulated_rates, float* simulated_rates0, float* accum_rates,
    const int pathN, int path,  
    float dtau = 0.5, float dt = 0.01)
{
    // calculated rate
    float rate;
    float sum_rate;

    // Simulation Parameters
    int stride = dtau / dt; // 
    const float sqrt_dt = sqrtf(dt);

    int t = threadIdx.x;
    int gindex = blockIdx.x * TIMEPOINTS + threadIdx.x;
    int rndIdx = blockIdx.x * pathN * VOL_DIM + path * VOL_DIM;

    // Normal variates
    float phi0 = rngNrmVar[rndIdx]; 
    float phi1 = rngNrmVar[rndIdx + 1]; 
    float phi2 = rngNrmVar[rndIdx + 2]; 

    // Evolve the whole curve from 0 to T ( 1:1 mapping t with threadIdx.x)
    if (t < TIMEPOINTS)
    {
        float dF = 0;
        if (t == (TIMEPOINTS - 1)) {
            dF = simulated_rates[gindex] - simulated_rates[gindex - 1];
        }
        else {
            dF = simulated_rates[gindex + 1] - simulated_rates[gindex];
        }

        rate = __musiela_sde2(
            drifts[t],
            volatilities[t],
            volatilities[TIMEPOINTS + t],
            volatilities[TIMEPOINTS * 2 + t],
            phi0,
            phi1,
            phi2,
            sqrt_dt,
            dF,
            simulated_rates[gindex],
            dtau,
            dt
        );

#ifdef HJM_PATH_SIMULATION_DEBUG
        printf("Path %d Block %d Thread %d index %d Forward Rate %f phi0 %f phi1 %f phi2 %f \n", path, blockIdx.x, threadIdx.x, gindex, rate, phi0, phi1, phi2);
#endif
        // accumulate rate for discount calculation
        sum_rate = accum_rates[gindex];
        sum_rate += rate;
        accum_rates[gindex] = sum_rate;

        // store the simulated rate
        simulated_rates0[gindex] = rate; //

        // update numeraire based on simulation block 
        if (path % stride == 0) {
            if (t == (path / stride)) {
                numeraires[gindex].x = rate;
                numeraires[gindex].y = __expf(-sum_rate * dt);
#ifdef HJM_NUMERAIRE_DEBUG
                //printf("Path %d Block %d Thread %d index %d Forward Rate %f Discount %f\n", path, blockIdx.x, threadIdx.x, gindex, rate, __expf(-sum_rate * dt));
#endif
            }
        }
    }
}



/*
 * Exposure generation kernel
 * one to one mapping between threadIdx.x and tenor
 * TODO - Multiply cashflow vector times triangular matrix to obtain the time to market and from there the expected exposure
 */
__global__
void _exposure_calc_kernel(float* exposure, float2* numeraires, const float notional, const float K, float* accrual, int simN)
{
    __shared__ float cash_flows[TIMEPOINTS];
    float discount_factor;
    float forward_rate;
    float cash_flow;
    float sum = 0.0;

    int globaltid = blockIdx.x * TIMEPOINTS + threadIdx.x;

    // calculate and load the cash flow in shared memory
    if (threadIdx.x < TIMEPOINTS) {
        forward_rate = numeraires[globaltid].x;
        discount_factor = numeraires[globaltid].y;   
        cash_flow = discount_factor * notional * accrual[threadIdx.x] * (forward_rate - K);
        cash_flows[threadIdx.x] = cash_flow;
#ifdef EXPOSURE_PROFILES_DEBUG
        printf("Block %d Thread %d Forward Rate %f Discount %f CashFlow %f \n", blockIdx.x, threadIdx.x, forward_rate, discount_factor, cash_flow);
#endif
    }
    __syncthreads();

#ifdef EXPOSURE_PROFILES_DEBUG
    if (threadIdx.x == 0) {
        for (int t = 0; t < TIMEPOINTS; t++) {
            printf("t - indext %d CashFlow %f \n", t, cash_flows[t]);
        }
    }
#endif

    // calculate the exposure profile
    if ( threadIdx.x < TIMEPOINTS )
    {
        for (int t = threadIdx.x + 1; t < TIMEPOINTS; t++) {
            sum += cash_flows[t];
        }
        sum = (sum > 0.0) ? sum : 0.0;
        exposure[globaltid] = sum;
#ifdef EXPOSURE_PROFILES_DEBUG
        printf("Block %d Thread %d Exposure %f \n", blockIdx.x, threadIdx.x, sum);
#endif
    }
    __syncthreads();
}



/*
* Calculate Expected Exposure Profile
* 2D Aggregation using cublas sgemv
*/

void __expectedexposure_calc_kernel(float* expected_exposure, float* exposures, float *d_x, float *d_y, hipblasHandle_t &handle, int exposureCount) {

    const float alpha = 1.f/(float) exposureCount;
    const float beta = 1.f;
    float cols = (float) TIMEPOINTS;
    float rows = (float) exposureCount;
    
    // Apply matrix x identity vector (all 1) to do a column reduction by rows
    CUBLAS_CALL ( hipblasSgemv(handle, HIPBLAS_OP_N, cols, rows,  &alpha, exposures, cols, d_x, 1, &beta, d_y, 1) );
    CUDA_RT_CALL( hipDeviceSynchronize() );

    CUDA_RT_CALL(hipMemcpy(expected_exposure, d_y, TIMEPOINTS * sizeof(float), hipMemcpyDeviceToHost));

#ifdef DEV_CURND_HOSTGEN1 
    printf("Exposure 2D Matrix Aggregation by Cols  \n");
    printf("Matrix Cols (%d) Rows(%d) x Vector (%d) in elapsed time %f ms \n", TIMEPOINTS, simN, simN, elapsed_time);
    printf("Effective Bandwidth: %f GB/s \n", 2 * TIMEPOINTS * simN * 4 / elapsed_time / 1e6);
#endif
}


/*
   Exposure Calculation Kernel Invocation
*/
void calculateExposureGPU(float* expected_exposure, InterestRateSwap payOff, float* accrual, float* spot_rates, float* drifts, float* volatilities, int exposureCount) {

    //int _simN = 32000; // 1000; // 1000; // 1000; // 256; // 100; 1024
    exposureCount = 5000;

    // HJM Model simulation number of paths with timestep dt = 0.01, dtau = 0.5 for 25 years
    const int pathN = 2500;
    const float dt = 0.01f;

    // Memory allocation 
    float* d_accrual = 0;
    float* d_spot_rates = 0;
    float* d_drifts = 0;
    float* d_volatilities = 0;
    float2* d_numeraire = 0;
    float* d_exposures = 0;
    float* simulated_rates = 0;
    float* simulated_rates0 = 0;
    float* accum_rates = 0;
    float* d_x = 0;;
    float* d_y = 0;

    
    // Select the GPU Device in a multigpu setup
    int gpu = 0;
    hipSetDevice(gpu);

    // Global memory reservation for constant input data
    CUDA_RT_CALL(hipMalloc((void**)&d_numeraire, exposureCount * TIMEPOINTS * sizeof(float2)));  // Numeraire (discount_factor, forward_rates)
    CUDA_RT_CALL(hipMalloc((void**)&d_exposures, exposureCount * TIMEPOINTS * sizeof(float)));   // Exposure profiles
    CUDA_RT_CALL(hipMalloc((void**)&simulated_rates, exposureCount * TIMEPOINTS * sizeof(float)));
    CUDA_RT_CALL(hipMalloc((void**)&simulated_rates0, exposureCount * TIMEPOINTS * sizeof(float)));
    CUDA_RT_CALL(hipMalloc((void**)&accum_rates, exposureCount * TIMEPOINTS * sizeof(float)));
    CUDA_RT_CALL(hipMemset(accum_rates, 0, exposureCount * TIMEPOINTS * sizeof(float)));
    CUDA_RT_CALL(hipMemset(simulated_rates0, 0, exposureCount * TIMEPOINTS * sizeof(float)));

    // Global memory reservation for constant input data
    CUDA_RT_CALL(hipMalloc((void**)&d_accrual, TIMEPOINTS * sizeof(float)));  // accrual
    CUDA_RT_CALL(hipMalloc((void**)&d_spot_rates, TIMEPOINTS * sizeof(float)));  // spot_rates
    CUDA_RT_CALL(hipMalloc((void**)&d_drifts, TIMEPOINTS * sizeof(float)));  // drifts
    CUDA_RT_CALL(hipMalloc((void**)&d_volatilities, VOL_DIM * TIMEPOINTS * sizeof(float)));  // volatilities

    // EE calculation aux vectors Global Memory (Convert to Const Memory)
    CUDA_RT_CALL(hipMalloc((void**)&d_x, exposureCount * sizeof(float)));
    CUDA_RT_CALL(hipMalloc((void**)&d_y, TIMEPOINTS * sizeof(float)));

    // Set Value for EE calculation aux vectors
    hipMemset(d_x, 1, exposureCount * sizeof(float));
    hipMemset(d_y, 0, TIMEPOINTS * sizeof(float));

    // CUBLAS handler
    hipblasHandle_t handle;
    CUBLAS_CALL (hipblasCreate(&handle));

    // Copy the spot_rates, drift & volatilities to device global memory Constant Memory TODO
    CUDA_RT_CALL(hipMemcpy(d_accrual, accrual, TIMEPOINTS * sizeof(float), hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(d_spot_rates, spot_rates, TIMEPOINTS * sizeof(float), hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(d_drifts, drifts, TIMEPOINTS * sizeof(float), hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(d_volatilities, volatilities, VOL_DIM * TIMEPOINTS * sizeof(float), hipMemcpyHostToDevice));

    // Global Memory reservation for RNG vector
#ifdef RNG_HOST_API
    float* rngNrmVar = 0;
    int rngCount = exposureCount * VOL_DIM * pathN;
    CUDA_RT_CALL(hipMalloc((void**)&rngNrmVar, rngCount * sizeof(float)));
#else
    const int rngCount = VOL_DIM * exposureCount;
    hiprandStateMRG32k3a* rngStates = 0;
    CUDA_RT_CALL(hipMalloc((void**)&rngStates, rngCount * sizeof(hiprandStateMRG32k3a)));
    CUDA_RT_CALL(hipDeviceSynchronize());
#endif

    // Random Number Generation 
    auto t_start = std::chrono::high_resolution_clock::now();
#ifdef RNG_HOST_API
    initRNG2_kernel(rngNrmVar, 1234ULL, rngCount);
#else
    int blockSize = 1024;
    int gridSize = (rngCount + blockSize - 1) / blockSize;;
    initRNG2_kernel << <gridSize, blockSize >> > (rngStates, 1234ULL, rngCount);
#endif
    auto t_end = std::chrono::high_resolution_clock::now();
    double elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << "total random normal variates " << rngCount << " generated in " << elapsed_time_ms << "(ms)" << std::endl;

    // Obtain number of SM per GPU device
    hipDeviceProp_t devprop;
    hipGetDeviceProperties(&devprop, gpu);
    //int sM = devprop.multiProcessorCount;

    // Initialize vectors already stored in Global Memory
    // simulated_rates values are initialized to the initial spot_rate values
    int blockSize = 64;
    int gridSize = exposureCount; // (exposureCount < sM) ? exposureCount : (exposureCount - sM + 1) / sM; // <- Grid Size exposureCount < 40 (Total number of SM RTX 2070)

    t_start = std::chrono::high_resolution_clock::now();

    initVectors_kernel<<< gridSize, blockSize >>>(d_numeraire, simulated_rates, d_spot_rates, exposureCount, dt);
    CUDA_RT_CALL(hipDeviceSynchronize());

    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << "total time to initialize data " << elapsed_time_ms << "(ms)" << std::endl;

#ifndef HJM_NUMERAIRE_DEBUG
    for (int s = 0; s < exposureCount; s++) {
        for (int t = 0; t < TIMEPOINTS; t++) {
            printf(" %f", simulated_rates[s * TIMEPOINTS + t]);
        }
    }
    std::cout << std::endl;
#endif

    // Monte Carlo Simulation kernel execution configuration 
    // Monte Carlos Simulation HJM Kernels (2500 paths)//dt = 0.01, dtau = 0.5
    blockSize = 64;
    gridSize = exposureCount; // 

    t_start = std::chrono::high_resolution_clock::now();

    for (int path = 1; path < pathN; path++)
    {
        __generatePaths_kernel <<< gridSize, blockSize >>> (
            d_numeraire,
            d_drifts,
            d_volatilities,
            rngNrmVar,
            simulated_rates,
            simulated_rates0,
            d_accrual,
            pathN,
            path
        );  
        CUDA_RT_CALL(hipDeviceSynchronize());

        // update simulated rates (swap pointers)
        std::swap(simulated_rates, simulated_rates0);
    }

    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << "total time taken to run all " << pathN * exposureCount << " HJM MC simulation " << elapsed_time_ms << "(ms)" << std::endl;

    // Exposure Profile Calculation 

    blockSize = 64;
    gridSize = exposureCount; // 

    t_start = std::chrono::high_resolution_clock::now();

    _exposure_calc_kernel <<<gridSize, blockSize>>>(d_exposures, d_numeraire, payOff.notional, payOff.K, d_accrual, exposureCount);
    CUDA_RT_CALL( hipDeviceSynchronize() );

    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << "total time taken to run all " << exposureCount << " exposure profile calculation " << elapsed_time_ms << "(ms)" << std::endl;

    // Expected Exposure Profile Calculation
    t_start = std::chrono::high_resolution_clock::now();

    __expectedexposure_calc_kernel(expected_exposure, d_exposures, d_x, d_y, handle, exposureCount);

    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << "total time taken to run" << exposureCount << " expected exposure profile " << elapsed_time_ms << "(ms)" << std::endl;

    // TODO - improve measurement GFLOPS

#ifdef EXPOSURE_PROFILES_AGGR_DEBUG
    printf("Expected Exposure Profile\n");
    for (int t = 0; t < TIMEPOINTS; t++) {
        printf("%1.4f ", expected_exposure[t]);
    }
    printf("\n");
#endif

    // Release Resources
    if (handle) {
        CUBLAS_CALL( hipblasDestroy(handle) );
    }

    if (d_x) {
        CUDA_RT_CALL(hipFree(d_x));
    }

    if (d_y) {
        CUDA_RT_CALL( hipFree(d_y));
    }

    if (d_numeraire) {
        CUDA_RT_CALL( hipFree(d_numeraire) );
    }
  
#ifdef RNG_HOST_API
    CUDA_RT_CALL(hipFree(rngNrmVar));
#else
    if (rngStates) {
        CUDA_RT_CALL(hipFree(rngStates));
    }
#endif

    if (d_accrual) {
        CUDA_RT_CALL(hipFree(d_accrual));
    }

    if (d_spot_rates) {
        CUDA_RT_CALL(hipFree(d_spot_rates));
    }

    if (d_drifts) {
        CUDA_RT_CALL(hipFree(d_drifts));
    }

    if (d_volatilities) {
        CUDA_RT_CALL(hipFree(d_volatilities));
    }

    if (d_exposures) {
        CUDA_RT_CALL(hipFree(d_exposures));
    }

    if (simulated_rates) {
        CUDA_RT_CALL(hipFree(simulated_rates));
    }

    if (simulated_rates0) {
        CUDA_RT_CALL(hipFree(simulated_rates0));
    }

    if (accum_rates) {
        CUDA_RT_CALL(hipFree(accum_rates));
    }
}



