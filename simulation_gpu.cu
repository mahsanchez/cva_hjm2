#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <omp.h>
#include <thread>
#include <thrust\device_vector.h>
#include "mkl.h"

#include <iostream>

#include "simulation_gpu.h"

#define FULL_MASK 0xffffffff
#define TILE_DIM 51
#define TIMEPOINTS 51
#define VOL_DIM 3
#define BLOCKSIZE 32
#define WARPSIZE 32
#define MAX_BLOCK_SZ 256
#define BATCH_SZ 1000

#undef HJM_SDE_DEBUG
#define MC_RDM_DEBUG
#undef HJM_PATH_SIMULATION_DEBUG
#undef HJM_NUMERAIRE_DEBUG
#undef EXPOSURE_PROFILES_DEBUG
#define DEV_CURND_HOSTGEN
#undef EXPOSURE_PROFILES_AGGR_DEBUG
#define EXPECTED_EXPOSURE_DEBUG
#define CONST_MEMORY
#define RNG_HOST_API
#undef RNG_DEV_API
#define UM_HINTS
#define TIME_COUNTERS

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus)                                                      \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
    }


#define TIMED_RT_CALL(x, y) \
{ \
    {auto t_start = std::chrono::high_resolution_clock::now(); \
    x; \
    auto t_end = std::chrono::high_resolution_clock::now(); \
    double elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count(); \
    printf("%s %f (ms) \n", y , elapsed_time_ms); }\
  \
} 


#define CURAND_CALL(x)                                 \
   {                                                   \
        if((x)!=HIPRAND_STATUS_SUCCESS)                 \
          printf("ERROR: CURAND call at %s:%d\n",__FILE__,__LINE__);\
                                                       \
    }  

#define CUBLAS_CALL(x)                                 \
   {                                                   \
        if((x)!=HIPBLAS_STATUS_SUCCESS)                 \
          printf("ERROR: CUBLAS call at %s:%d\n",__FILE__,__LINE__);\
                                                       \
    } 


#ifdef CONST_MEMORY
    __constant__ float d_accrual[TIMEPOINTS];
    __constant__ float d_spot_rates[TIMEPOINTS];
    __constant__ float d_drifts[TIMEPOINTS];
    __constant__ float d_volatilities[VOL_DIM * TIMEPOINTS];
#endif


/*
 * MarketData Struct
*/
struct MarketData {
    float* accrual;
    float* spot_rates;
    float* drifts;
    float* volatilities;
};


/*
 * Musiela Parametrization SDE
 * We simulate the SDE f(t+dt)=f(t) + dfbar  
 * where SDE dfbar =  m(t)*dt+SUM(Vol_i*phi[i]*SQRT(dt))+dF/dtau*dt and phi ~ N(0,1)
 */

__device__
inline float __musiela_sde2(float drift, float vol0, float vol1, float vol2, float phi0, float phi1, float phi2, float sqrt_dt, float dF, float rate0, float dtau, float dt) {
    float vol_sum = vol0 * phi0;
    vol_sum += vol1 * phi1;
    vol_sum += vol2 * phi2;
    vol_sum *= sqrtf(dt);

    float dfbar = drift * dt;
    dfbar += vol_sum;

    dfbar += (dF / dtau) * dt;

    // apply Euler Maruyana
    float result = rate0 + dfbar;

    return result;
}


/**
* * RNG init Kernel
*/

#ifdef RNG_HOST_API
void initRNG2_kernel(float* rngNrmVar, const unsigned int seed, int rnd_count)
{
    const float mean = 0.0;  const float stddev = 1.0;
    hiprandGenerator_t generator;
    CURAND_CALL( hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT) );
    CURAND_CALL( hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL) );
    CURAND_CALL( hiprandGenerateNormal(generator, rngNrmVar, rnd_count, mean, stddev) );
    CUDA_RT_CALL( hipDeviceSynchronize() );
    CURAND_CALL(hiprandDestroyGenerator(generator));
}

void initRNG2_kernel(float* rngNrmVar, const unsigned int seed, int rnd_count, const float mean, const float stddev)
{
    hiprandGenerator_t generator;
    CURAND_CALL(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, seed));
    CURAND_CALL(hiprandGenerateNormal(generator, rngNrmVar, rnd_count, mean, stddev));
    CUDA_RT_CALL(hipDeviceSynchronize());
    CURAND_CALL(hiprandDestroyGenerator(generator));
}
#else
__global__ void initRNG2_kernel(hiprandStateMRG32k3a* const rngStates, const unsigned int seed, int rnd_count)
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    for (; index < rnd_count; index += blockDim.x * gridDim.x) {
        hiprand_init(seed, index, 0, &rngStates[index]);
    }
}
#endif




/*
 * Monte Carlo HJM Path Generation Constant Memory
*/
__global__
void __generatePaths_kernel(float2* numeraires, 
    /*float* d_spot_rates,
    float* d_drifts, float* d_volatilities, */ 
    void* rngNrmVar,
    float* simulated_rates, float* simulated_rates0, float* accum_rates,
    const int pathN, int path,  
    float dtau = 0.5, float dt = 0.01)
{
    // calculated rate
    float rate;
    float sum_rate;

    // Simulation Parameters
    int stride = dtau / dt; // 
    const float sqrt_dt = sqrtf(dt);

    int t = threadIdx.x;
    int gindex = blockIdx.x * TIMEPOINTS + threadIdx.x;

#ifdef RNG_HOST_API
    float phi0;
    float phi1;
    float phi2;
#else
    __shared__ float phi0;
    __shared__ float phi1;
    __shared__ float phi2;
#endif

    // Evolve the whole curve from 0 to T ( 1:1 mapping t with threadIdx.x)
    if (t < TIMEPOINTS)
    {
        if (path == 0) {
            rate = d_spot_rates[t];
        }
        else {
            // Calculate dF term in Musiela Parametrization SDE
            float dF = 0;
            if (t == (TIMEPOINTS - 1)) {
                dF = simulated_rates[gindex] - simulated_rates[gindex - 1];
            }
            else {
                dF = simulated_rates[gindex + 1] - simulated_rates[gindex];
            }

            // Normal random variates
 #ifdef RNG_HOST_API
            float *rngNrms = (float*)rngNrmVar;
            int rndIdx = blockIdx.x * pathN * VOL_DIM + path * VOL_DIM;
            phi0 = rngNrms[rndIdx];
            phi1 = rngNrms[rndIdx + 1];
            phi2 = rngNrms[rndIdx + 2];
#else
            if (threadIdx.x == 0) {
                hiprandStateMRG32k3a *state = (hiprandStateMRG32k3a*) rngNrmVar;
                hiprandStateMRG32k3a localState = state[blockIdx.x];
                phi0 = hiprand_uniform(&localState);
                phi1 = hiprand_uniform(&localState);
                phi2 = hiprand_uniform(&localState);
                state[blockIdx.x] = localState;
            }
            __syncthreads();         
#endif

            // simulate the sde
            rate = __musiela_sde2(
                d_drifts[t],
                d_volatilities[t],
                d_volatilities[TIMEPOINTS + t],
                d_volatilities[TIMEPOINTS * 2 + t],
                phi0,
                phi1,
                phi2,
                sqrt_dt,
                dF,
                simulated_rates[gindex],
                dtau,
                dt
            );
        }

#ifdef HJM_PATH_SIMULATION_DEBUG
        printf("Path %d Block %d Thread %d index %d Forward Rate %f phi0 %f phi1 %f phi2 %f \n", path, blockIdx.x, threadIdx.x, gindex, rate, phi0, phi1, phi2);
#endif
        // accumulate rate for discount calculation
        sum_rate = accum_rates[gindex];
        sum_rate += rate;
        accum_rates[gindex] = sum_rate;

        // store the simulated rate
        simulated_rates0[gindex] = rate; //

        // update numeraire based on simulation block 
        if (path % stride == 0) {
            if (t == (path / stride)) {
                numeraires[gindex].x = rate;
                numeraires[gindex].y = __expf(-sum_rate * dt);
#ifdef HJM_NUMERAIRE_DEBUG
                printf("Path %d Block %d Thread %d index %d Forward Rate %f Discount %f\n", path, blockIdx.x, threadIdx.x, gindex, rate, __expf(-sum_rate * dt));
#endif
            }
        }
    }
}


/*
 * Monte Carlo HJM Path Generation
*/
    __global__
        void __generatePaths_kernel(
            float *d_spot_rates,
            float* d_drifts, 
            float* d_volatilities, 
            float2* numeraires,            
            void* rngNrmVar,
            float* simulated_rates, float* simulated_rates0, float* accum_rates,
            const int pathN, int path,
            float dtau = 0.5, float dt = 0.01)
    {
        // calculated rate
        float rate;
        float sum_rate;

        // Simulation Parameters
        int stride = dtau / dt; // 
        const float sqrt_dt = sqrtf(dt);

        int t = threadIdx.x;
        int gindex = blockIdx.x * TIMEPOINTS + threadIdx.x;

#ifdef RNG_HOST_API
        float phi0;
        float phi1;
        float phi2;
#else
        __shared__ float phi0;
        __shared__ float phi1;
        __shared__ float phi2;
#endif

        // Evolve the whole curve from 0 to T ( 1:1 mapping t with threadIdx.x)
        if (t < TIMEPOINTS)
        {
            if (path == 0) {
                rate = d_spot_rates[t];
            }
            else {
                // Calculate dF term in Musiela Parametrization SDE
                float dF = 0;
                if (t == (TIMEPOINTS - 1)) {
                    dF = simulated_rates[gindex] - simulated_rates[gindex - 1];
                }
                else {
                    dF = simulated_rates[gindex + 1] - simulated_rates[gindex];
                }

                // Normal random variates
#ifdef RNG_HOST_API
                float* rngNrms = (float*)rngNrmVar;
                int rndIdx = blockIdx.x * pathN * VOL_DIM + path * VOL_DIM;
                phi0 = rngNrms[rndIdx];
                phi1 = rngNrms[rndIdx + 1];
                phi2 = rngNrms[rndIdx + 2];
#else
                if (threadIdx.x == 0) {
                    hiprandStateMRG32k3a* state = (hiprandStateMRG32k3a*)rngNrmVar;
                    hiprandStateMRG32k3a localState = state[blockIdx.x];
                    phi0 = hiprand_uniform(&localState);
                    phi1 = hiprand_uniform(&localState);
                    phi2 = hiprand_uniform(&localState);
                    state[blockIdx.x] = localState;
                }
                __syncthreads();
#endif

                // simulate the sde
                rate = __musiela_sde2(
                    d_drifts[t],
                    d_volatilities[t],
                    d_volatilities[TIMEPOINTS + t],
                    d_volatilities[TIMEPOINTS * 2 + t],
                    phi0,
                    phi1,
                    phi2,
                    sqrt_dt,
                    dF,
                    simulated_rates[gindex],
                    dtau,
                    dt
                );
            }

#ifdef HJM_PATH_SIMULATION_DEBUG
            printf("Path %d Block %d Thread %d index %d Forward Rate %f phi0 %f phi1 %f phi2 %f \n", path, blockIdx.x, threadIdx.x, gindex, rate, phi0, phi1, phi2);
#endif
            // accumulate rate for discount calculation
            sum_rate = accum_rates[gindex];
            sum_rate += rate;
            accum_rates[gindex] = sum_rate;

            // store the simulated rate
            simulated_rates0[gindex] = rate; //

            // update numeraire based on simulation block 
            if (path % stride == 0) {
                if (t == (path / stride)) {
                    numeraires[gindex].x = rate;
                    numeraires[gindex].y = __expf(-sum_rate * dt);
#ifdef HJM_NUMERAIRE_DEBUG
                    printf("Path %d Block %d Thread %d index %d Forward Rate %f Discount %f\n", path, blockIdx.x, threadIdx.x, gindex, rate, __expf(-sum_rate * dt));
#endif
                }
            }
        }
    }


/*
* Risk Factor Generation 
*/

void riskFactorSim(
    int gridSize, int blockSize, 
    MarketData marketData,
    float2* numeraires,
    void* rngNrmVar,
    float* simulated_rates, 
    float* simulated_rates0, 
    float* accum_rates,
    const int pathN, 
    float dtau = 0.5, 
    float dt = 0.01)
{

    for (int path = 0; path < pathN; path++)
    {
        __generatePaths_kernel <<< gridSize, blockSize >>> (
            marketData.spot_rates,
            marketData.drifts,
            marketData.volatilities,
            numeraires,
            rngNrmVar,
            simulated_rates,
            simulated_rates0,
            accum_rates,
            pathN,
            path,
            dtau,
            dt
            );

        CUDA_RT_CALL(hipDeviceSynchronize());

        // update simulated rates (swap pointers)
        std::swap(simulated_rates, simulated_rates0);
    }
}


/*
 * Exposure generation kernel
 * one to one mapping between threadIdx.x and tenor
 */
__global__
void _exposure_calc_kernel(float* exposure, float2* numeraires, const float notional, const float K, /*float* d_accrual,*/ int simN, float dtau = 0.5f)
{
    __shared__ float cash_flows[TIMEPOINTS];
    float discount_factor;
    float forward_rate;
    float libor;
    float cash_flow;
    float sum = 0.0;
    float m = (1.0f / dtau);

    int globaltid = blockIdx.x * TIMEPOINTS + threadIdx.x;

    // calculate and load the cash flow in shared memory
    if (threadIdx.x < TIMEPOINTS) {
        forward_rate = numeraires[globaltid].x;
        libor = m * (__expf(forward_rate/m) - 1.0f);
        discount_factor = numeraires[globaltid].y;   
        cash_flow = discount_factor * notional * d_accrual[threadIdx.x] * (libor - K);
        cash_flows[threadIdx.x] = cash_flow;
#ifdef EXPOSURE_PROFILES_DEBUG
        printf("Block %d Thread %d Forward Rate %f libor %f Discount %f CashFlow %f \n", blockIdx.x, threadIdx.x, forward_rate, libor, discount_factor, cash_flow);
#endif
    }
    __syncthreads();

#ifdef EXPOSURE_PROFILES_DEBUG
    if (threadIdx.x == 0) {
        for (int t = 0; t < TIMEPOINTS; t++) {
            printf("t - indext %d CashFlow %f \n", t, cash_flows[t]);
        }
    }
#endif

    // calculate the exposure profile
    if ( threadIdx.x < TIMEPOINTS )
    {
        for (int t = threadIdx.x + 1; t < TIMEPOINTS; t++) {
            sum += cash_flows[t];
        }
        sum = (sum > 0.0) ? sum : 0.0;
        exposure[globaltid] = sum;
#ifdef EXPOSURE_PROFILES_DEBUG
        printf("Block %d Thread %d Exposure %f \n", blockIdx.x, threadIdx.x, sum);
#endif
    }
    __syncthreads();
}



/*
* Calculate Expected Exposure Profile
* 2D Aggregation using cublas sgemv
*/
void __expectedexposure_calc_kernel(float* expected_exposure, float* exposures, float *d_x, float *d_y, hipblasHandle_t handle, int exposureCount) {

    const float alpha = 1.f / (float)exposureCount;
    const float beta = 1.f ;
    float cols = (float) TIMEPOINTS;
    float rows = (float) exposureCount;
    
    // Apply matrix x identity vector (all 1) to do a column reduction by rows
    CUBLAS_CALL ( hipblasSgemv(handle, HIPBLAS_OP_N, cols, rows,  &alpha, exposures, cols, d_x, 1, &beta, d_y, 1) );
    CUDA_RT_CALL( hipMemcpy(expected_exposure, d_y, TIMEPOINTS * sizeof(float), hipMemcpyDeviceToHost));

#ifdef DEV_CURND_HOSTGEN1 
    printf("Exposure 2D Matrix Aggregation by Cols  \n");
    printf("Matrix Cols (%d) Rows(%d) x Vector (%d) in elapsed time %f ms \n", TIMEPOINTS, simN, simN, elapsed_time);
    printf("Effective Bandwidth: %f GB/s \n", 2 * TIMEPOINTS * simN * 4 / elapsed_time / 1e6);
#endif
}


/*
   Exposure Calculation Kernel Invocation
*/
void calculateExposureGPU(float* expected_exposure, InterestRateSwap payOff, float* accrual, float* spot_rates, float* drifts, float* volatilities, int exposureCount, float dt) {

    //exposureCount = 5000; // change exposure count here for testing 5, 10, 1000, 5000, 10000, 20000, 50000

    // HJM Model simulation number of paths with timestep dt = 0.01, expiry = 25 years
    const int pathN = payOff.expiry / dt; // 2500

    // Memory allocation 
#ifndef CONST_MEMORY
    float* d_accrual = 0;
    float* d_spot_rates = 0;
    float* d_drifts = 0;
    float* d_volatilities = 0;
#endif
    float2* d_numeraire = 0;
    float* d_exposures = 0;
    float* simulated_rates = 0;
    float* simulated_rates0 = 0;
    float* accum_rates = 0;
    float* d_x = 0;;
    float* d_y = 0;

    
    // Select the GPU Device in a multigpu setup
    int gpu = 0;
    hipSetDevice(gpu);

    // Global memory reservation for constant input data
    CUDA_RT_CALL(hipMalloc((void**)&d_numeraire, exposureCount * TIMEPOINTS * sizeof(float2)));  // Numeraire (discount_factor, forward_rates)
    CUDA_RT_CALL(hipMalloc((void**)&d_exposures, exposureCount * TIMEPOINTS * sizeof(float)));   // Exposure profiles
    CUDA_RT_CALL(hipMalloc((void**)&simulated_rates, exposureCount * TIMEPOINTS * sizeof(float)));
    CUDA_RT_CALL(hipMalloc((void**)&simulated_rates0, exposureCount * TIMEPOINTS * sizeof(float)));
    CUDA_RT_CALL(hipMalloc((void**)&accum_rates, exposureCount * TIMEPOINTS * sizeof(float)));

    // Global memory reservation for constant input data
#ifndef CONST_MEMORY
    CUDA_RT_CALL(hipMalloc((void**)&d_accrual, TIMEPOINTS * sizeof(float)));  // accrual
    CUDA_RT_CALL(hipMalloc((void**)&d_spot_rates, TIMEPOINTS * sizeof(float)));  // spot_rates
    CUDA_RT_CALL(hipMalloc((void**)&d_drifts, TIMEPOINTS * sizeof(float)));  // drifts
    CUDA_RT_CALL(hipMalloc((void**)&d_volatilities, VOL_DIM * TIMEPOINTS * sizeof(float)));  // volatilities
#endif

    // EE calculation aux vectors Global Memory (Convert to Const Memory)
    CUDA_RT_CALL(hipMalloc((void**)&d_x, exposureCount * sizeof(float)));
    CUDA_RT_CALL(hipMalloc((void**)&d_y, TIMEPOINTS * sizeof(float)));

    // initialize accum_rates float array
    int N = exposureCount * TIMEPOINTS;
    thrust::device_ptr<float> dev_ptr(accum_rates);
    thrust::fill(dev_ptr, dev_ptr + N, (float) 0.0f);

    // initialize simulated_rates0 float array
    thrust::device_ptr<float> dev_ptr2(simulated_rates0);
    thrust::fill(dev_ptr2, dev_ptr2 + N, (float)0.0f);

    // initialize d_x float array
    N = exposureCount;
    thrust::device_ptr<float> dev_ptr3(d_x);
    thrust::fill(dev_ptr3, dev_ptr3 + N, (float) 1.0f);

    // initialize d_x float array
    N = TIMEPOINTS;
    thrust::device_ptr<float> dev_ptr4(d_y);
    thrust::fill(dev_ptr4, dev_ptr4 + N, (float)0.0f);

    // CUBLAS handler
    hipblasHandle_t handle;
    CUBLAS_CALL (hipblasCreate(&handle));

#ifdef CONST_MEMORY
    CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_accrual), accrual, TIMEPOINTS * sizeof(float)));
    CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_spot_rates), spot_rates, TIMEPOINTS * sizeof(float)));
    CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_drifts), drifts, TIMEPOINTS * sizeof(float)));
    CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_volatilities), volatilities, VOL_DIM * TIMEPOINTS * sizeof(float)));
#else
    CUDA_RT_CALL(hipMemcpy(d_accrual, accrual, TIMEPOINTS * sizeof(float), hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(d_spot_rates, spot_rates, TIMEPOINTS * sizeof(float), hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(d_drifts, drifts, TIMEPOINTS * sizeof(float), hipMemcpyHostToDevice));
    CUDA_RT_CALL(hipMemcpy(d_volatilities, volatilities, VOL_DIM * TIMEPOINTS * sizeof(float), hipMemcpyHostToDevice));
#endif

    // Global Memory reservation for RNG vector
#ifdef RNG_HOST_API
    float* rngNrmVar = 0;
    int rngCount = exposureCount * VOL_DIM * pathN;
    CUDA_RT_CALL(hipMalloc((void**)&rngNrmVar, rngCount * sizeof(float)));
#else
    const int rngCount = exposureCount;
    hiprandStateMRG32k3a* rngNrmVar = 0;
    CUDA_RT_CALL(hipMalloc((void**)&rngNrmVar, rngCount * sizeof(hiprandStateMRG32k3a)));
    CUDA_RT_CALL(hipDeviceSynchronize());
#endif

    // kernel dimension variables
    int blockSize;
    int gridSize;

    // Random Number Generation 
    auto t_start = std::chrono::high_resolution_clock::now();
#ifdef RNG_HOST_API
    initRNG2_kernel(rngNrmVar, 1234ULL, rngCount);
#else
    blockSize = 32;
    gridSize = (rngCount + blockSize - 1) / blockSize;;
    initRNG2_kernel << <gridSize, blockSize >> > (rngNrmVar, 1234ULL, rngCount);
#endif
    auto t_end = std::chrono::high_resolution_clock::now();
    double elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << "total random normal variates " << rngCount << " generated in " << elapsed_time_ms << "(ms)" << std::endl;

    // Obtain number of SM per GPU device
    hipDeviceProp_t devprop;
    hipGetDeviceProperties(&devprop, gpu);
    //int sM = devprop.multiProcessorCount;

    // Risk Factor Generation by using Monte Carlo Simulation (HJM Framework / Musiela SDE)
    // Monte Carlos Simulation HJM Grid (2500 paths)//dt = 0.01, dtau = 0.5, expiry = 25
    blockSize = 64;
    gridSize = exposureCount; // (exposureCount < sM) ? exposureCount : (exposureCount - sM + 1) 

    t_start = std::chrono::high_resolution_clock::now();

    for (int path = 0; path < pathN; path++)
    {
        __generatePaths_kernel <<< gridSize, blockSize >>> (
            d_numeraire,
            rngNrmVar,
            simulated_rates,
            simulated_rates0,
            accum_rates,
            pathN,
            path,
            payOff.dtau,
            dt
        );  
        CUDA_RT_CALL(hipDeviceSynchronize());

        // update simulated rates (swap pointers)
        std::swap(simulated_rates, simulated_rates0);
    }

    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << "total time taken to run all " << pathN * exposureCount << " HJM MC simulation " << elapsed_time_ms << "(ms)" << std::endl;

    // Exposure Profile Calculation 

    blockSize = 64;
    gridSize = exposureCount; // 

    t_start = std::chrono::high_resolution_clock::now();

    _exposure_calc_kernel <<<gridSize, blockSize>>>(d_exposures, d_numeraire, payOff.notional, payOff.K, /*d_accrual,*/ exposureCount);
    CUDA_RT_CALL( hipDeviceSynchronize() );

    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << "total time taken to run all " << exposureCount << " exposure profile calculation " << elapsed_time_ms << "(ms)" << std::endl;


#ifdef EXPOSURE_PROFILES_AGGR_DEBUG
    float* exposures = (float*)malloc(exposureCount * TIMEPOINTS * sizeof(float));

    CUDA_RT_CALL(hipMemcpy(exposures, d_exposures, exposureCount * TIMEPOINTS * sizeof(float), hipMemcpyDeviceToHost));

    printf("Exposure Profile\n");
    for (int s = 0; s < exposureCount; s++) {
        for (int t = 0; t < TIMEPOINTS; t++) {
            printf("%1.4f ", exposures[s * TIMEPOINTS + t]);
        }
        printf("\n");
    }
    
    free(exposures);
#endif

    // Expected Exposure Profile Calculation
    t_start = std::chrono::high_resolution_clock::now();

    float* result = (float* ) malloc( TIMEPOINTS * sizeof(float));

    __expectedexposure_calc_kernel(result, d_exposures, d_x, d_y, handle, exposureCount);

    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << "total time taken to run" << exposureCount << " expected exposure profile " << elapsed_time_ms << "(ms)" << std::endl;

    // TODO - improve measurement GFLOPS

#ifdef EXPECTED_EXPOSURE_DEBUG
    printf("Expected Exposure Profile\n");
    for (int t = 0; t < TIMEPOINTS; t++) {
        printf("%1.4f ", result[t]);
    }
    printf("\n");
#endif

    free(result);

    // Release Resources
    if (handle) {
        CUBLAS_CALL( hipblasDestroy(handle) );
    }

    if (d_x) {
        CUDA_RT_CALL(hipFree(d_x));
    }

    if (d_y) {
        CUDA_RT_CALL( hipFree(d_y));
    }

    if (d_numeraire) {
        CUDA_RT_CALL( hipFree(d_numeraire) );
    }
  
    if (rngNrmVar) {
        CUDA_RT_CALL(hipFree(rngNrmVar));
    }

#ifndef CONST_MEMORY
    if (d_accrual) {
        CUDA_RT_CALL(hipFree(d_accrual));
    }

    if (d_spot_rates) {
        CUDA_RT_CALL(hipFree(d_spot_rates));
    }

    if (d_drifts) {
        CUDA_RT_CALL(hipFree(d_drifts));
    }

    if (d_volatilities) {
        CUDA_RT_CALL(hipFree(d_volatilities));
    }
#endif

    if (d_exposures) {
        CUDA_RT_CALL(hipFree(d_exposures));
    }

    if (simulated_rates) {
        CUDA_RT_CALL(hipFree(simulated_rates));
    }

    if (simulated_rates0) {
        CUDA_RT_CALL(hipFree(simulated_rates0));
    }

    if (accum_rates) {
        CUDA_RT_CALL(hipFree(accum_rates));
    }
}


/*
   Exposure Calculation Kernel Invocation
*/
void calculateExposureMultiGPU(float* expected_exposure, InterestRateSwap payOff, float* accrual, float* spot_rates, float* drifts, float* volatilities, int scenarios, float dt) {

    const int num_gpus = 4;
    //hipGetDeviceCount(&num_gpus);

    float* rngNrmVar[num_gpus];
    const int pathN = payOff.expiry / dt; // 25Y requires 2500 simulations
    int scenarios_gpus = scenarios / num_gpus; // total work distribution across gpus
    int rnd_count = scenarios_gpus * VOL_DIM * pathN;
    const unsigned int seed = 1234ULL;
    const float mean = 0.0;
    const float stddev = 1.0;

    // flattern the market data
    float* data = 0;
    int totalMarketDataSize = ( 3 * TIMEPOINTS + 3 * TIMEPOINTS * VOL_DIM ) * sizeof(float);
    hipMallocManaged( &data, totalMarketDataSize );
    // copy accrual, spot_rates, drifts, volatilites to marketData
    MarketData marketData{ data, data + TIMEPOINTS, data + 2 * TIMEPOINTS, data + 3 * TIMEPOINTS };

    // intermediate & final results memory reservation on device data
    float2* d_numeraire[num_gpus];
    float* d_exposures[num_gpus];
    float* simulated_rates[num_gpus];
    float* simulated_rates0[num_gpus];
    float* accum_rates[num_gpus];
    float* d_x[num_gpus];
    float* d_y[num_gpus];
    float* partial_exposure[num_gpus];

    // memory allocation
    for (int gpuDevice = 0; gpuDevice < num_gpus; gpuDevice++) {

        hipSetDevice(gpuDevice);

        CUDA_RT_CALL(hipMalloc((void**)&rngNrmVar[gpuDevice], rnd_count * sizeof(float)));
        CUDA_RT_CALL(hipMalloc((void**)&d_numeraire[gpuDevice], scenarios_gpus * TIMEPOINTS * sizeof(float2)));  // Numeraire (discount_factor, forward_rates)
        CUDA_RT_CALL(hipMalloc((void**)&d_exposures[gpuDevice], scenarios_gpus * TIMEPOINTS * sizeof(float)));   // Exposure profiles
        CUDA_RT_CALL(hipMalloc((void**)&simulated_rates[gpuDevice], scenarios_gpus * TIMEPOINTS * sizeof(float)));
        CUDA_RT_CALL(hipMalloc((void**)&simulated_rates0[gpuDevice], scenarios_gpus * TIMEPOINTS * sizeof(float)));
        CUDA_RT_CALL(hipMalloc((void**)&accum_rates[gpuDevice], scenarios_gpus * TIMEPOINTS * sizeof(float)));
        CUDA_RT_CALL(hipMalloc((void**)&d_x[gpuDevice], scenarios_gpus * sizeof(float)));
        CUDA_RT_CALL(hipMalloc((void**)&d_y[gpuDevice], TIMEPOINTS * sizeof(float)));
        partial_exposure[num_gpus] = (float *) malloc(TIMEPOINTS * sizeof(float));

        // initialize accum_rates float array
        int N = scenarios_gpus * TIMEPOINTS;
        thrust::device_ptr<float> dev_ptr(accum_rates[gpuDevice]);
        thrust::fill(dev_ptr, dev_ptr + N, (float) 0.0f);

        // initialize simulated_rates0 float array
        thrust::device_ptr<float> dev_ptr2(simulated_rates0[gpuDevice]);
        thrust::fill(dev_ptr2, dev_ptr2 + N, (float) 0.0f);

        // initialize d_x float array
        thrust::device_ptr<float> dev_ptr3(d_x[gpuDevice]);
        thrust::fill(dev_ptr3, dev_ptr3 + scenarios_gpus, (float)1.0f);

        // initialize d_x float array
        thrust::device_ptr<float> dev_ptr4(d_y[gpuDevice]);
        thrust::fill(dev_ptr4, dev_ptr4 + TIMEPOINTS, (float)0.0f);
    }

    // HJM Simulation Kernel Execution Parameters
    int blockSize = 64;
    int gridSize = scenarios_gpus;

    omp_set_num_threads(num_gpus);
    #pragma omp parallel
    {
        int gpuDevice = omp_get_thread_num();

        // printf("1: %d Thread# %d: x = %d\n", omp_get_num_threads(), omp_get_thread_num(), 2);
        hipSetDevice(gpuDevice);

        // create Random Numbers (change seed by adding the gpuDevice)
        TIMED_RT_CALL(
            initRNG2_kernel(rngNrmVar[gpuDevice], seed, rnd_count, mean, stddev),  "normal variate generation"
        );

        // Read-only data is duplicated and accessed locally. The data is made available to all GPUs by prefetching.
#ifdef UM_HINTS
        CUDA_RT_CALL(hipMemAdvise(data, totalMarketDataSize, hipMemAdviseSetReadMostly, 0));
#endif 

        // Prefetching here causes read duplication of data instead of data migration
        CUDA_RT_CALL(hipMemPrefetchAsync(data, totalMarketDataSize, gpuDevice));

        // Risk Factor Simulations  
        TIMED_RT_CALL( 
            riskFactorSim(
                blockSize,
                gridSize, 
                marketData,
                d_numeraire[gpuDevice],
                rngNrmVar[gpuDevice],
                simulated_rates[gpuDevice],
                simulated_rates0[gpuDevice],
                accum_rates[gpuDevice],
                pathN,
                payOff.dtau,
                dt
            ),
            "Total Execution Time HJM MC simulation"
        );

        // Exposure Profile Calculation  TODO (d_exposures + gpuDevice * TIMEPOINTS)
        _exposure_calc_kernel <<< gridSize, blockSize >>> (d_exposures[gpuDevice], d_numeraire[gpuDevice], payOff.notional, payOff.K, scenarios / num_gpus);

        // Partial Expected Exposure Calculation and scattered across gpus
        hipblasHandle_t handle; CUBLAS_CALL(hipblasCreate(&handle));
        TIMED_RT_CALL(
             __expectedexposure_calc_kernel(partial_exposure[gpuDevice], d_exposures[gpuDevice], d_x[gpuDevice], d_y[gpuDevice], handle, scenarios / num_gpus),
            "partial expected exposure profile"
        );

        // sum up the exposure profile  using a critical section, declare the exposure profile_vector as a shared openmp variable
        #pragma omp critical
        vsAdd(TIMEPOINTS, expected_exposure, partial_exposure[gpuDevice], expected_exposure);

        // free up resources
        if (handle) {
            CUBLAS_CALL(hipblasDestroy(handle));
        }
    }

    // free up resources
    if (data) {
        CUDA_RT_CALL(hipFree(data));
    }

    for (int gpuDevice = 0; gpuDevice < num_gpus; gpuDevice++) {

        hipSetDevice(gpuDevice);

        if (rngNrmVar[gpuDevice]) {
            CUDA_RT_CALL(hipFree(rngNrmVar[gpuDevice]));
        }

        if (d_numeraire[gpuDevice]) {
            CUDA_RT_CALL(hipFree(d_numeraire[gpuDevice]));
        }

        if (d_exposures[gpuDevice]) {
            CUDA_RT_CALL(hipFree(d_exposures[gpuDevice]));
        }

        if (simulated_rates[gpuDevice]) {
            CUDA_RT_CALL(hipFree(simulated_rates[gpuDevice]));
        }

        if (simulated_rates0[gpuDevice]) {
            CUDA_RT_CALL(hipFree(simulated_rates0[gpuDevice]));
        }

        if (accum_rates[gpuDevice]) {
            CUDA_RT_CALL(hipFree(accum_rates[gpuDevice]));
        }

        if (d_x[gpuDevice]) {
            CUDA_RT_CALL(hipFree(d_x[gpuDevice]));
        }

        if (d_y[gpuDevice]) {
            CUDA_RT_CALL(hipFree(d_y[gpuDevice]));
        }
    }
}
